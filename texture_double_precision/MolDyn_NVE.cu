#include "hip/hip_runtime.h"
#include "MolDyn_NVE.h"

//#define equilibration

using namespace std;

int main(int argc, char** argv){

  int tentativo = 1;
  temp = 0.8;
  m_temp=0;
  int N;
  accettazione = 0.001;
  double errore = abs(m_temp-temp); 
  Particles P;
  Input(&P); 

  //assign symbol to costant memory
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_npart), &npart, sizeof(int)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_box), &box, sizeof(double)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_binsize), &bin_size, sizeof(double)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_delta), &delta, sizeof(double)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_rcut), &rcut, sizeof(double)));

#ifdef equilibration
  cout << "equilibration phase! " << endl;
  while ( errore > accettazione ) { //until equilibration
        cout <<"################################################################" << endl;
cout << "                tentativo numero: " << tentativo << endl;
        cout <<"################################################################" << endl;
        if (tentativo == 1)
                restart = 0;
        else
                restart = 1;

#else
  if (argc!=2) {cerr << "insert restart parameter" << endl;
              return -1;}

  restart = atoi(argv[1]);
#endif
  cout << "restart = " << restart << endl;

  P.TotalTime = 0;
  Initialization(&P);  //Initialize initial configurations

  Measure(&P);

  cout << "Initial potential energy (with tail corrections) = " << stima_pot_gpu << endl;
  cout << "Pressure (with tail corrections) = " << stima_press_gpu << endl;   
  cout << "Ekin = " << stima_kin_gpu << endl << endl;  //questa deve venire 1.2, perchè ho riscalato le velocità per avere
					              // temperatura = 0.8

  HANDLE_ERROR( hipEventCreate( &P.start ));
  HANDLE_ERROR( hipEventCreate( &P.stop ));
  HANDLE_ERROR( hipEventRecord( P.start, 0 ) );
  cout<< "\n\n";

  N = 100; //number of blocks for data_blocking analysis

 //doing MD steps
  for(int istep=1; istep <= nstep; ++istep) {
     Move_gpu(&P); //move with verlet-algorithm
     if (istep%10 == 0) Measure(&P); //measure physical properties
     if (istep%iprint == 0) cout << "Number of time-steps: " << istep << endl; 
  }
  
  //save instantaneous results on file
  print_properties(); 
  //doing data analysis and save on file the results
  data_blocking_MD(N); 
  
  //error between mean_temperature obtained from data analysis and target temperature
  errore = abs(m_temp-temp);
  cout << "ora l'errore tra la temperatura del sistema e quella target è: " << errore << endl;
  //save target configuration in old.0 and old.final
  system ("make copy");
  //overwrite config.0 and config.final
  ConfFinal(&P);   

  HANDLE_ERROR( hipEventRecord( P.stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( P.stop ) );
  HANDLE_ERROR( hipEventElapsedTime( &P.TotalTime,
                                        P.start, P.stop ) );
  ofstream Time("simulation.time",ios::app);
  Time << npart << "\t" << P.TotalTime << endl;
  Time.close();
  printf( "Time:  %3.1f ms\n",P.TotalTime);

#ifdef equilibration
  tentativo++;
  }
#else
  cout << endl;
  cout <<"################################################################" << endl;
  cout << "REMEMBER: if want to save final and penultimate configurations" << endl;
  cout <<"in file old.0 (last one) and old.final(penultimate) do command-> make copy" << endl;
  cout <<"##################################################################" << endl;
  cout << endl;
#endif
  exit(&P);

  return 0;
}
