#include "hip/hip_runtime.h"
#include "MolDyn_NVE.h"

//#define equilibration

using namespace std;

int main(int argc, char** argv){
//to control if our device support map memory, unified memory

    /*hipDeviceProp_t  prop;
    int whichDevice;
    HANDLE_ERROR( hipGetDevice( &whichDevice ) );
    HANDLE_ERROR( hipGetDeviceProperties( &prop, whichDevice ) );
    if (prop.canMapHostMemory != 1) {
        printf( "Device can not map memory.\n" );
        return 0;
    }
   HANDLE_ERROR( hipSetDeviceFlags( hipDeviceMapHost ) );
 */

  hipSetDevice(1);
  int tentativo = 1;
  temp = 0.8;
  m_temp=0;
  int N=100;
  accettazione = 0.001;
  float errore = abs(m_temp-temp); 
  Particles P;
  Input(&P); 

  //assign symbol to costant memory
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_npart), &npart, sizeof(int)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_box), &box, sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_binsize), &bin_size, sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_delta), &delta, sizeof(float)));
  HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(gpu_rcut), &rcut, sizeof(float)));

#ifdef equilibration
  cout << "equilibration phase! " << endl;
  while ( errore > accettazione ) { //until equilibration
        cout <<"################################################################" << endl;
cout << "                tentativo numero: " << tentativo << endl;
        cout <<"################################################################" << endl;
        if (tentativo == 1)
                restart = 1;
        else
                restart = 1;

#else
  if (argc!=2) {cerr << "insert restart parameter" << endl;
              return -1;}

  restart = atoi(argv[1]);
#endif
  cout << "restart = " << restart << endl;

  P.TotalTime = 0;
  Initialization(&P);  //Initialize initial configurations

  Measure(&P);

  cout << "Initial potential energy (with tail corrections) = " << stima_pot_gpu << endl;
  cout << "Pressure (with tail corrections) = " << stima_press_gpu << endl;   
  cout << "Ekin = " << stima_kin_gpu << endl << endl;  //questa deve venire 1.2, perchè ho riscalato le velocità per avere
					              // temperatura = 0.8
  cout << "starting measuring time" << endl;
  HANDLE_ERROR( hipEventCreate( &P.start ));
  HANDLE_ERROR( hipEventCreate( &P.stop ));
  HANDLE_ERROR( hipEventRecord( P.start, 0 ) );
  cout<< "\n\n";
 //doing MD steps
  for(int istep=1; istep <= nstep; ++istep) {
     Move_gpu(&P); //move with verlet-algorithm
     if (istep%10 == 0) Measure(&P); //measure physical properties
     if (istep%iprint == 0) cout << "Number of time-steps: " << istep << endl; 
  }
  cout << "stop measuring" << endl;
  HANDLE_ERROR( hipEventRecord( P.stop, 0 ) );
  HANDLE_ERROR( hipEventSynchronize( P.stop ) );
  HANDLE_ERROR( hipEventElapsedTime( &P.TotalTime,
                                        P.start, P.stop ) );
 
  //save instantaneous results on file
  print_properties(); 
  //doing data analysis and save on file the results
  data_blocking_MD(N); 
  
  //error between mean_temperature obtained from data analysis and target temperature
  errore = abs(m_temp-temp);
  cout << "ora l'errore tra la temperatura del sistema e quella target è: " << errore << endl;
  //save target configuration in old.0 and old.final
  system ("make copy");
  //overwrite config.0 and config.final
  ConfFinal(&P);   

  ofstream Time("simulation.time",ios::app);
  Time << npart << "\t" << P.TotalTime << endl;
  Time.close();
  printf( "Time:  %3.1f ms\n",P.TotalTime);

#ifdef equilibration
  tentativo++;
  }
#else
  cout << endl;
  cout <<"################################################################" << endl;
  cout << "REMEMBER: if want to save final and penultimate configurations" << endl;
  cout <<"in file old.0 (last one) and old.final(penultimate) do command-> make copy" << endl;
  cout <<"##################################################################" << endl;
  cout << endl;
#endif
  exit(&P);

  return 0;
}
